#include "hip/hip_runtime.h"
#include "CannyGPU.cuh"
#include "SM_Loader.cuh"

// CUDA global constants
__constant__ Kernel_weights const_weights;

__global__ void sobel_kernel(unsigned char* img, unsigned char* sobel_module, float* sobel_dir, unsigned char kernel_size, unsigned char radius,
                             unsigned int sizeSM, short rows, short cols, unsigned char L2_norm);

__global__ void non_max_suppresion_kernel(unsigned char* sobel_module, float* sobel_dir, unsigned char* out,
                                          unsigned int sizeSM, unsigned char radius, short rows, short cols);

__global__ void hysteresis_kernel(unsigned char* img_non_max_sup, unsigned char* out, unsigned char* sobel_module,
                                  unsigned int sizeSM, unsigned char radius, short rows, short cols, int low_tr, int high_tr);

void CannyGPU(unsigned char* img_host,
              unsigned char* out,
              short rows,
              short cols,
              unsigned char kernel_size,
              int low_tr,
              int high_tr,
              unsigned char L2_norm){

    unsigned char *out_non_max_device, *out_device;
    float *sobel_dir_device;
    unsigned char *img_device, *sobel_module_device;
    unsigned int factor=16;
    unsigned int size;
    int sizeSMbyte;
    unsigned char radius;


    dim3 num_blocks, num_threads_per_block;
    // Kernel config.
    num_threads_per_block.y=factor;
    num_threads_per_block.x=factor;
    //
    num_blocks.y = rows/num_threads_per_block.y+((rows%num_threads_per_block.y)==0? 0:1);
    num_blocks.x = cols/num_threads_per_block.x+((cols%num_threads_per_block.x)==0? 0:1);

    // image allocation size.
    size=rows*cols*sizeof(unsigned char);

    // Data allocation on device.
    hipMalloc((void**)&img_device, size);
    hipMalloc((void**)&sobel_dir_device, rows*cols*sizeof(float));
    hipMalloc((void**)&sobel_module_device, size);
    hipMalloc((void**)&out_non_max_device, size);
    hipMalloc((void**)&out_device, size);

    // Copy data from host to device.
    hipMemcpy(img_device, img_host, size, hipMemcpyHostToDevice);

    // Defines the kernel weights.
    Kernel_weights k;
    init_kernel_weights(k, kernel_size);
    // Copy the struct kernel weights to Constant Memory.
    hipMemcpyToSymbol(HIP_SYMBOL(const_weights), &k, sizeof(k));

    // Calculate the radius of convolution radius.
    radius=int(floor((kernel_size-1)/2));

    // Calculate the size of the SM needed.
    sizeSMbyte = (num_threads_per_block.y+kernel_size-1)*(num_threads_per_block.x+kernel_size-1)*sizeof(unsigned char);
    unsigned int sizeSM = (num_threads_per_block.x+kernel_size-1);


    // Sobel kernel
    sobel_kernel<<<num_blocks, num_threads_per_block, sizeSMbyte>>>(img_device, sobel_module_device, sobel_dir_device, kernel_size, radius, sizeSM, rows, cols, L2_norm);
    hipDeviceSynchronize();

    // NMS kernel
    sizeSMbyte = (num_threads_per_block.x+3-1)*(num_threads_per_block.y+3-1)*sizeof(unsigned char);
    sizeSM = (num_threads_per_block.x+3-1); // per questa fase viene utilizzato sempre un kernel 3x3
    radius=1; // kernel 3x3 -> quindi radius=1
    non_max_suppresion_kernel<<<num_blocks, num_threads_per_block, sizeSMbyte>>>(sobel_module_device, sobel_dir_device, out_non_max_device, sizeSM, radius, rows, cols);
    hipDeviceSynchronize();
    // Hysteresis kernel
    sizeSMbyte = 2*(num_threads_per_block.x+3-1)*(num_threads_per_block.y+3-1)*sizeof(unsigned char);
    hysteresis_kernel<<<num_blocks, num_threads_per_block, sizeSMbyte>>>(out_non_max_device, out_device, sobel_module_device, sizeSM, radius, rows, cols, low_tr, high_tr);
    hipDeviceSynchronize();

    // Copy the resulting array from the device to the host.
    hipMemcpy(out, out_device, size, hipMemcpyDeviceToHost);

    // Free the memory from the device.
    hipFree(out_non_max_device);
    hipFree(out_device);
    hipFree(img_device);
    hipFree(sobel_module_device);
    hipFree(sobel_dir_device);

}


__global__ void hysteresis_kernel(unsigned char* img_non_max_sup,
                                  unsigned char* out,
                                  unsigned char* sobel_module,
                                  unsigned int sizeSM,
                                  unsigned char radius,
                                  short rows,
                                  short cols,
                                  int low_tr,
                                  int high_tr){

    /*
    The SM is a contiguous memory area, so if you need to allocate multiple arrays ...
    you have to allocate them in a contiguous manner.
    For example, if I need to allocate two arrays; after allocating the first,
    just define a pointer to the memory location immediately following the memory address
    of the last element of the first array, and start allocating from that location onwards.
    */
    extern __shared__ unsigned char sm[];
    unsigned char* sm_mag = &sm[0];
    unsigned char* sm_non_max = &sm[sizeSM*sizeSM];

    // Load the magnitude of the pixels that fall into the block from GM to SM.
    SM_data_loader(sm_mag, sobel_module, sizeSM, radius, rows, cols);
    // Load the value of the pixels of the matrix resulting from the NMS phase that fall into the block from GM to SM.
    SM_data_loader(sm_non_max, img_non_max_sup, sizeSM, radius, rows, cols);

    // Synchronize all threads in the block to make sure all threads have finished writing within the SM.
    __syncthreads();

    // Compute the global indexes of the thread.
    unsigned int y = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;

    // if the thread "falls" inside the image
    if(y<rows && x<cols){

        ////
        // I remember that you have to add a shift of + radius to the x and y coordinates when accessing the SM ...
        // to make sure that the thread index inside the block corresponds to the pixel index inside the SM matrix.
        ////
        int local_index = (threadIdx.y+radius)*sizeSM+(threadIdx.x+radius);

        // if in the previous phase the pixel has been discarded, it cannot be "imaged",
        // and therefore 0 is assigned in the final matrix.
        if(sm_non_max[local_index]==0) out[y*cols+x]=0;
        // The pixels that are edges -> coming from the NMS phase are considered.
        else{

            bool edge = false;
            // If the magnitude of the pixel is greater than the high tr, then it is a "strong" edge.
            if(sm_mag[local_index]>high_tr) edge=true;
            // If the pixel magnitude is less than the low tr, then the edge is discarded.
            else if(sm_mag[local_index]<low_tr) edge=false;
            // If the magnitude of the pixel is between the two thresholds,
            // then it is considered as a valid edge only if it is in an 8-connected neighborhood of a "strong" edge.
            else if(sm_mag[local_index]>=low_tr && sm_mag[local_index]<=high_tr){

                for(int i=0; i<3; i++)
                    for(int j=0; j<3; j++){

                        // If a pixel that is in a neighborhood 8-connected to
                        // the considered "eligible" edge pixel is a "strong" edge, then it is also considered as a valid edge pixel.
                        if(sm_mag[(threadIdx.y+i)*sizeSM+(threadIdx.x+j)]>high_tr){
                              edge=true;
                              // Trick to get out of the double for loop
                              i=j=3;
                          }
                    }
            }

            // Mark whether it is an edge pixel or not.
            if(edge) out[y*cols+x]=255;
              else out[y*cols+x]=0;

        }
    }
}

__global__ void non_max_suppresion_kernel(unsigned char* sobel_module,
                                          float* sobel_dir,
                                          unsigned char* out,
                                          unsigned int sizeSM,
                                          unsigned char radius,
                                          short rows,
                                          short cols){

    // Define shared memory data
    extern __shared__ unsigned char sm[];
    // Load the magnitude of the pixels that fall into the block from GM to SM.
    SM_data_loader(sm, sobel_module, sizeSM, radius, rows, cols);

    // Synchronize all threads in the block to make sure all threads have finished writing within the SM.
    __syncthreads();

    // Compute the global indexes of the thread.
    unsigned int y = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;

    // Take from the GM la the orientation of the gradient of the pixel under consideration.
    float currDir = sobel_dir[y*cols+x];
    ////
    // I remember that you have to add a + radius shift to the x and y coordinates when accessing the SM ...
    // to make sure that the thread index inside the block matches the pixel index inside the SM matrix.
    ////
    unsigned char mag = sm[(threadIdx.y+radius)*sizeSM+(threadIdx.x+radius)];
    // Normalize the orientation of the gradient.
    while(currDir<0) currDir+=180;

    bool check=true;

    if(y>=rows-1 || y<=0 || x>=cols-1 || x<=0) check=false;
    else{
        // The edges are always located in a direction orthogonal to that of the gradient;
        // Reason for which, we consider the direction orthogonal to that of the gradient in the desired pixel ...
        // and if in this orientation the magnitude of the pixel is greater than that of the two adjacent pixels,
        //then it is considered as an edge pixel.

        if(currDir>22.5 && currDir<=67.5){
            if(mag<sm[(threadIdx.y-1 +radius)*sizeSM+(threadIdx.x-1 +radius)] ||
               mag<sm[(threadIdx.y+1 +radius)*sizeSM+(threadIdx.x+1 +radius)]) check = false;
        }

        else if(currDir>67.5 && currDir<=112.5){
            if(mag<sm[(threadIdx.y-1 +radius)*sizeSM+(threadIdx.x +radius)] ||
               mag<sm[(threadIdx.y+1 +radius)*sizeSM+(threadIdx.x +radius)]) check = false;

        }

        else if(currDir>112.5 && currDir<=157.5){
            if(mag<sm[(threadIdx.y+1 +radius)*sizeSM+(threadIdx.x-1 +radius)] ||
              mag<sm[(threadIdx.y-1 +radius)*sizeSM+(threadIdx.x+1 +radius)]) check = false;

        }

        else{
            if(mag<sm[(threadIdx.y +radius)*sizeSM+(threadIdx.x-1 +radius)] ||
              mag<sm[(threadIdx.y +radius)*sizeSM+(threadIdx.x+1 +radius)]) check = false;
        }

    }
    if(check) out[y*cols+x]=255;
    else out[y*cols+x]=0;

}


__global__ void sobel_kernel(unsigned char* img,
                             unsigned char* sobel_module,
                             float* sobel_dir,
                             unsigned char kernel_size,
                             unsigned char radius,
                             unsigned int sizeSM,
                             short rows,
                             short cols,
                             unsigned char L2_norm){

    // Define shared memory data
    extern __shared__ unsigned char sm[];

    // Upload the necessary data from GM to SM.
    SM_data_loader(sm, img, sizeSM, radius, rows, cols);

    // Synchronize all threads in the block to make sure all threads have finished writing within the SM.
    __syncthreads();

    // Conv step

    // Calculate the derivative with respect to x and y.
    float sumX=0, sumY=0;
    for (int y=0; y<kernel_size; y++)
        for (int x=0; x<kernel_size; x++){
          sumY += sm[(threadIdx.y+y)*sizeSM+(threadIdx.x+x)]*const_weights.y[y][x];
          sumX += sm[(threadIdx.y+y)*sizeSM+(threadIdx.x+x)]*const_weights.x[y][x];
        }

    // Compute the global indexes of the thread.
    unsigned int y = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int x = blockIdx.x*blockDim.y+threadIdx.x;

    // This check is used to verify that the thread is "contained" within the image.
    if (y<rows && x<cols){

      // Calculate the magnitude of the gredient by applying the norm2 or an approximation of it using the abs.
      int pixel_intensity;
      if(L2_norm==0){
          pixel_intensity = abs(sumY) + abs(sumX);
      }
      else{
          pixel_intensity = sqrt((sumY*sumY)+(sumX*sumX));
      }
      pixel_intensity = pixel_intensity > 255? 255: pixel_intensity < 0? 0: pixel_intensity;

      // Access the GM and save the pixel magnitude value.
      sobel_module[y*cols+x] = pixel_intensity;

      // Calculate the orientation of the gradient, such as arctang (dy / dx).
      sobel_dir[y*cols+x] = atan2(sumY,sumX)*(180/M_PI);

    }
}


void init_kernel_weights(Kernel_weights &k, unsigned char kernel_size){

    memset(&k, 0, sizeof(k));

    if(kernel_size==3){
        k.y[0][0]=1; k.y[0][1]=2; k.y[0][2]=1;
        k.y[1][0]=0; k.y[1][1]=0; k.y[1][2]=0;
        k.y[2][0]=-1; k.y[2][1]=-2; k.y[2][2]=-1;
        //
        k.x[0][0]=1; k.x[0][1]=0; k.x[0][2]=-1;
        k.x[1][0]=2; k.x[1][1]=0; k.x[1][2]=-2;
        k.x[2][0]=1; k.x[2][1]=0; k.x[2][2]=-1;
    }
    else if(kernel_size==5){

    }
    else if(kernel_size==7){

    }


}
