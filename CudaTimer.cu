#include "CudaTimer.cuh"

CudaTimer::CudaTimer(){
    hipEventCreate(&start);
    hipEventCreate(&stop);
    time=0;
}

CudaTimer::~CudaTimer(){
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void CudaTimer::start_timer(){
    hipEventRecord(start, 0);
}

void CudaTimer::stop_timer(){
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
}

float CudaTimer::get_time(){

    return time;
}
